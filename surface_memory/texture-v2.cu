#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>

#define WIDTH 2048
#define HEIGHT 2048

// Kernel using texture object (read-only)
__global__ void advectTextureKernel(hipTextureObject_t texObj, float *output,
                                    int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    float value = tex2D<float>(texObj, x + 0.5f, y + 0.5f);
    output[y * width + x] = value;
  }
}

// Kernel using surface object (read & write)
__global__ void advectSurfaceKernel(hipSurfaceObject_t surfObj, int width,
                                    int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    float value;
    surf2Dread(&value, surfObj, x * sizeof(float), y);
    // For demonstration: write the same value back
    surf2Dwrite(value, surfObj, x * sizeof(float), y);
  }
}

void checkCuda(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << "Error: " << msg << ": " << hipGetErrorString(err)
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main() {
  size_t size = WIDTH * HEIGHT * sizeof(float);

  // Allocate host memory
  float *h_input = new float[WIDTH * HEIGHT];
  for (int i = 0; i < WIDTH * HEIGHT; ++i) {
    h_input[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Allocate device memory
  float *d_output;
  checkCuda(hipMalloc(&d_output, size), "hipMalloc d_output");

  // Create CUDA array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray_t cuArray;
  checkCuda(hipMallocArray(&cuArray, &channelDesc, WIDTH, HEIGHT,
                            hipArraySurfaceLoadStore),
            "hipMallocArray");

  // Copy host data to CUDA array
  checkCuda(hipMemcpy2DToArray(cuArray, 0, 0, h_input, WIDTH * sizeof(float),
                                WIDTH * sizeof(float), HEIGHT,
                                hipMemcpyHostToDevice),
            "hipMemcpy2DToArray");

  // Create texture object
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(texRes));
  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cuArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t texObj = 0;
  checkCuda(hipCreateTextureObject(&texObj, &texRes, &texDesc, nullptr),
            "hipCreateTextureObject");

  // Create surface object
  hipSurfaceObject_t surfObj = 0;
  hipResourceDesc surfRes;
  memset(&surfRes, 0, sizeof(surfRes));
  surfRes.resType = hipResourceTypeArray;
  surfRes.res.array.array = cuArray;
  checkCuda(hipCreateSurfaceObject(&surfObj, &surfRes),
            "hipCreateSurfaceObject");

  // Setup execution parameters
  dim3 threads(16, 16);
  dim3 blocks((WIDTH + threads.x - 1) / threads.x,
              (HEIGHT + threads.y - 1) / threads.y);

  // Warm-up runs
  advectTextureKernel<<<blocks, threads>>>(texObj, d_output, WIDTH, HEIGHT);
  hipDeviceSynchronize();
  advectSurfaceKernel<<<blocks, threads>>>(surfObj, WIDTH, HEIGHT);
  hipDeviceSynchronize();

  // Timing: Texture
  hipEvent_t startTex, stopTex;
  hipEventCreate(&startTex);
  hipEventCreate(&stopTex);
  hipEventRecord(startTex, 0);
  advectTextureKernel<<<blocks, threads>>>(texObj, d_output, WIDTH, HEIGHT);
  hipEventRecord(stopTex, 0);
  hipEventSynchronize(stopTex);
  float texTime = 0.0f;
  hipEventElapsedTime(&texTime, startTex, stopTex);

  // Timing: Surface
  hipEvent_t startSurf, stopSurf;
  hipEventCreate(&startSurf);
  hipEventCreate(&stopSurf);
  hipEventRecord(startSurf, 0);
  advectSurfaceKernel<<<blocks, threads>>>(surfObj, WIDTH, HEIGHT);
  hipEventRecord(stopSurf, 0);
  hipEventSynchronize(stopSurf);
  float surfTime = 0.0f;
  hipEventElapsedTime(&surfTime, startSurf, stopSurf);

  // Copy result back to host
  float *h_output = new float[WIDTH * HEIGHT];
  checkCuda(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost),
            "hipMemcpy output");

  // Print results
  std::cout << "Texture + Global Memory Kernel Time: " << texTime << " ms"
            << std::endl;
  std::cout << "Surface Memory Kernel Time: " << surfTime << " ms" << std::endl;

  // Cleanup
  hipDestroyTextureObject(texObj);
  hipDestroySurfaceObject(surfObj);
  hipFreeArray(cuArray);
  hipFree(d_output);
  delete[] h_input;
  delete[] h_output;

  hipEventDestroy(startTex);
  hipEventDestroy(stopTex);
  hipEventDestroy(startSurf);
  hipEventDestroy(stopSurf);

  return 0;
}
