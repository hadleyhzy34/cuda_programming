#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>

#define WIDTH 2048
#define HEIGHT 2048
#define ITERATIONS 10

// Kernel using texture object + global memory (pseudo one-pass via copy-back)
__global__ void advectTextureKernel(hipTextureObject_t texObj, float *output,
                                    int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    float value = tex2D<float>(texObj, x + 0.5f, y + 0.5f);
    // Simple update: add small constant
    value += 0.001f;
    output[y * width + x] = value;
  }
}

// Kernel using surface object (true one-pass)
__global__ void advectSurfaceKernel(hipSurfaceObject_t surfObj, int width,
                                    int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    float value;
    surf2Dread(&value, surfObj, x * sizeof(float), y);
    // Simple update: add small constant
    value += 0.001f;
    surf2Dwrite(value, surfObj, x * sizeof(float), y);
  }
}

void checkCuda(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << "Error: " << msg << ": " << hipGetErrorString(err)
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main() {
  size_t size = WIDTH * HEIGHT * sizeof(float);

  // Allocate host memory
  float *h_input = new float[WIDTH * HEIGHT];
  for (int i = 0; i < WIDTH * HEIGHT; ++i) {
    h_input[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Allocate device output buffer (texture variant)
  float *d_output;
  checkCuda(hipMalloc(&d_output, size), "hipMalloc d_output");

  // Create CUDA array with surface load/store enabled
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray_t cuArray;
  checkCuda(hipMallocArray(&cuArray, &channelDesc, WIDTH, HEIGHT,
                            hipArraySurfaceLoadStore),
            "hipMallocArray");

  // Copy host data to CUDA array
  checkCuda(hipMemcpy2DToArray(cuArray, 0, 0, h_input, WIDTH * sizeof(float),
                                WIDTH * sizeof(float), HEIGHT,
                                hipMemcpyHostToDevice),
            "hipMemcpy2DToArray");

  // Create texture object
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(texRes));
  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cuArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t texObj = 0;
  checkCuda(hipCreateTextureObject(&texObj, &texRes, &texDesc, nullptr),
            "hipCreateTextureObject");

  // Create surface object
  hipSurfaceObject_t surfObj = 0;
  hipResourceDesc surfRes;
  memset(&surfRes, 0, sizeof(surfRes));
  surfRes.resType = hipResourceTypeArray;
  surfRes.res.array.array = cuArray;
  checkCuda(hipCreateSurfaceObject(&surfObj, &surfRes),
            "hipCreateSurfaceObject");

  dim3 threads(16, 16);
  dim3 blocks((WIDTH + threads.x - 1) / threads.x,
              (HEIGHT + threads.y - 1) / threads.y);

  // Benchmark Texture: iterative with copy-back
  hipEvent_t startTex, stopTex;
  hipEventCreate(&startTex);
  hipEventCreate(&stopTex);
  hipEventRecord(startTex, 0);
  for (int iter = 0; iter < ITERATIONS; iter++) {
    advectTextureKernel<<<blocks, threads>>>(texObj, d_output, WIDTH, HEIGHT);
    checkCuda(hipDeviceSynchronize(), "Kernel launch (Texture)");

    // Copy result from output buffer back to array
    checkCuda(hipMemcpy2DToArray(cuArray, 0, 0, d_output,
                                  WIDTH * sizeof(float), WIDTH * sizeof(float),
                                  HEIGHT, hipMemcpyDeviceToDevice),
              "CopyBack Texture");
  }
  hipEventRecord(stopTex, 0);
  hipEventSynchronize(stopTex);
  float texTime = 0.0f;
  hipEventElapsedTime(&texTime, startTex, stopTex);

  // Benchmark Surface: iterative in-place update
  hipEvent_t startSurf, stopSurf;
  hipEventCreate(&startSurf);
  hipEventCreate(&stopSurf);
  hipEventRecord(startSurf, 0);
  for (int iter = 0; iter < ITERATIONS; iter++) {
    advectSurfaceKernel<<<blocks, threads>>>(surfObj, WIDTH, HEIGHT);
    checkCuda(hipDeviceSynchronize(), "Kernel launch (Surface)");
  }
  hipEventRecord(stopSurf, 0);
  hipEventSynchronize(stopSurf);
  float surfTime = 0.0f;
  hipEventElapsedTime(&surfTime, startSurf, stopSurf);

  // Print results
  std::cout << "Texture (One-Pass) Time: " << texTime << " ms" << std::endl;
  std::cout << "Surface (One-Pass) Time: " << surfTime << " ms" << std::endl;

  // Cleanup
  hipDestroyTextureObject(texObj);
  hipDestroySurfaceObject(surfObj);
  hipFreeArray(cuArray);
  hipFree(d_output);
  delete[] h_input;

  hipEventDestroy(startTex);
  hipEventDestroy(stopTex);
  hipEventDestroy(startSurf);
  hipEventDestroy(stopSurf);

  return 0;
}
