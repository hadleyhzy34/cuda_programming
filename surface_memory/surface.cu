#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <cuda_surface_types.h>
#include <>
#include <iostream>
#include <surface_functions.h>
#include <vector>

class SurfaceMemoryManager {
private:
  hipArray_t cuda_array;
  hipSurfaceObject_t surface_object;
  hipTextureObject_t texture_object; // For comparison

public:
  // Initialize 2D surface memory
  void initializeSurface2D(int width, int height) {
    hipError_t err;
    // 1. Create CUDA array descriptor
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();

    // 2. Allocate CUDA array and future bind surface
    hipMallocArray(&cuda_array, &channel_desc, width, height,
                    hipArraySurfaceLoadStore);

    // Create surface objects
    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;

    surface_object = 0;
    res_desc.res.array.array = cuda_array;
    err = hipCreateSurfaceObject(&surface_object, &res_desc);
    if (err != hipSuccess) {
      std::cerr << "hipCreateSurfaceObject failed for input: "
                << hipGetErrorString(err) << "\n";
      hipFreeArray(cuda_array);
      return;
    }

    // 3. Create texture object
    hipResourceDesc resource_desc = {};
    resource_desc.resType = hipResourceTypeArray;
    resource_desc.res.array.array = cuda_array;

    // 4. Optional: Create texture object for read operations
    hipTextureDesc texture_desc = {};
    texture_desc.addressMode[0] = hipAddressModeClamp;
    texture_desc.addressMode[1] = hipAddressModeClamp;
    texture_desc.filterMode = hipFilterModeLinear;
    texture_desc.readMode = hipReadModeElementType;
    texture_desc.normalizedCoords = 0;

    hipCreateTextureObject(&texture_object, &resource_desc, &texture_desc,
                            nullptr);
  }

  // Initialize 3D surface memory
  void initializeSurface3D(int width, int height, int depth) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();

    hipExtent extent = make_hipExtent(width, height, depth);
    hipMalloc3DArray(&cuda_array, &channel_desc, extent,
                      hipArraySurfaceLoadStore);

    hipResourceDesc resource_desc = {};
    resource_desc.resType = hipResourceTypeArray;
    resource_desc.res.array.array = cuda_array;

    hipCreateSurfaceObject(&surface_object, &resource_desc);
  }

  hipSurfaceObject_t getSurface() { return surface_object; }
  hipTextureObject_t getTexture() { return texture_object; }
  hipArray_t getArray() { return cuda_array; }

  ~SurfaceMemoryManager() {
    hipDestroySurfaceObject(surface_object);
    hipDestroyTextureObject(texture_object);
    hipFreeArray(cuda_array);
  }
};

// ============================================================================
// SURFACE MEMORY ACCESS PATTERNS
// ============================================================================

// Basic surface read/write operations
__global__ void surfaceBasicOperations(hipSurfaceObject_t surface, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    // READ from surface
    float value;
    surf2Dread(&value, surface, x * sizeof(float), y);

    // WRITE to surface
    float new_value = value * 2.0f + 1.0f;
    surf2Dwrite(new_value, surface, x * sizeof(float), y);
  }
}

// Advanced: Gaussian blur using surface memory
__global__ void gaussianBlurSurface(hipSurfaceObject_t input_surface,
                                    hipSurfaceObject_t output_surface,
                                    int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  // printf("thread execution on %d,%d\n", x, y);

  // 5x5 Gaussian kernel
  const float kernel[5][5] = {
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {7 / 273.0f, 26 / 273.0f, 41 / 273.0f, 26 / 273.0f, 7 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}};

  float sum = 0.0f;

  // Convolution with automatic boundary handling
  for (int dy = -2; dy <= 2; dy++) {
    for (int dx = -2; dx <= 2; dx++) {
      // Surface not automatically handles boundary conditions
      if (x + dx >= 0 && x + dx < width && y + dy >= 0 && y + dy < height) {
        float pixel;
        surf2Dread(&pixel, input_surface, (x + dx) * sizeof(float), y + dy);
        sum += pixel * kernel[dy + 2][dx + 2];
      }
    }
  }

  surf2Dwrite(sum, output_surface, x * sizeof(float), y);
}

// ============================================================================
// PERFORMANCE COMPARISON: GLOBAL vs TEXTURE vs SURFACE
// ============================================================================

// Global memory version (baseline)
__global__ void gaussianBlurGlobal(float *input, float *output, int width,
                                   int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const float kernel[5][5] = {
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {7 / 273.0f, 26 / 273.0f, 41 / 273.0f, 26 / 273.0f, 7 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}};

  float sum = 0.0f;

  for (int dy = -2; dy <= 2; dy++) {
    for (int dx = -2; dx <= 2; dx++) {
      int nx = x + dx;
      int ny = y + dy;

      // Manual boundary checking
      nx = max(0, min(width - 1, nx));
      ny = max(0, min(height - 1, ny));

      sum += input[ny * width + nx] * kernel[dy + 2][dx + 2];
    }
  }

  output[y * width + x] = sum;
}

// Texture memory version (read-only)
__global__ void gaussianBlurTexture(hipTextureObject_t input_texture,
                                    float *output, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const float kernel[5][5] = {
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {7 / 273.0f, 26 / 273.0f, 41 / 273.0f, 26 / 273.0f, 7 / 273.0f},
      {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
      {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}};

  float sum = 0.0f;

  // printf("handling thread: %d, %d\n", x, y);

  for (int dy = -2; dy <= 2; dy++) {
    for (int dx = -2; dx <= 2; dx++) {
      // Hardware interpolation and boundary handling
      float pixel = tex2D<float>(input_texture, x + dx + 0.5f, y + dy + 0.5f);
      sum += pixel * kernel[dy + 2][dx + 2];
    }
  }

  output[y * width + x] = sum;
}

// ============================================================================
// BEST PRACTICES AND OPTIMIZATION TECHNIQUES
// ============================================================================

// Best Practice 1: Use proper memory layout
class OptimizedSurfaceProcessor {
private:
  hipArray_t cuda_array;
  hipSurfaceObject_t surface;
  hipStream_t stream;

public:
  void setupOptimizedSurface(int width, int height) {
    // Use proper channel format for your data
    hipChannelFormatDesc desc;

    // For single precision float
    desc = hipCreateChannelDesc<float>();

    // For multi-channel data (RGBA)
    // desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

    // Allocate with optimal flags
    hipMallocArray(&cuda_array, &desc, width, height,
                    hipArraySurfaceLoadStore);

    // Create surface with resource descriptor
    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cuda_array;

    hipCreateSurfaceObject(&surface, &res_desc);

    // Create dedicated stream for surface operations
    hipStreamCreate(&stream);
  }

  // Best Practice 2: Optimal kernel launch configuration
  void launchOptimizedKernel(int width, int height) {
    // Use 2D block dimensions that align with hardware
    dim3 block_size(16, 16); // 256 threads per block (optimal for most GPUs)
    dim3 grid_size((width + block_size.x - 1) / block_size.x,
                   (height + block_size.y - 1) / block_size.y);

    // Launch with stream for better concurrency
    gaussianBlurSurface<<<grid_size, block_size, 0, stream>>>(surface, surface,
                                                              width, height);
  }
};

// Best Practice 3: Memory coalescing with surface memory
__global__ void coalescedSurfaceAccess(hipSurfaceObject_t surface, int width,
                                       int height) {
  // Calculate global thread position
  int global_x = blockIdx.x * blockDim.x + threadIdx.x;
  int global_y = blockIdx.y * blockDim.y + threadIdx.y;

  // Use shared memory to reduce surface accesses
  __shared__ float shared_data[18][18]; // 16x16 + 2 border

  int local_x = threadIdx.x + 1;
  int local_y = threadIdx.y + 1;

  // Load main data
  if (global_x < width && global_y < height) {
    surf2Dread(&shared_data[local_y][local_x], surface,
               global_x * sizeof(float), global_y);
  }

  // Load border data (edge threads load extra elements)
  if (threadIdx.x == 0 && global_x > 0) {
    surf2Dread(&shared_data[local_y][0], surface,
               (global_x - 1) * sizeof(float), global_y);
  }
  // ... similar for other borders

  __syncthreads();

  // Now process using shared memory instead of repeated surface reads
  if (global_x < width && global_y < height) {
    float result = 0.0f;
    for (int dy = -1; dy <= 1; dy++) {
      for (int dx = -1; dx <= 1; dx++) {
        result += shared_data[local_y + dy][local_x + dx] * 0.111f;
      }
    }

    surf2Dwrite(result, surface, global_x * sizeof(float), global_y);
  }
}

// ============================================================================
// PERFORMANCE BENCHMARKING
// ============================================================================

class SurfaceMemoryBenchmark {
public:
  void runBenchmarks(int width, int height, int iterations) {
    std::cout << "=== CUDA Surface Memory Benchmark ===" << std::endl;
    std::cout << "Image size: " << width << "x" << height << std::endl;
    std::cout << "Iterations: " << iterations << std::endl << std::endl;

    // Setup data
    std::vector<float> host_data(width * height);
    for (int i = 0; i < width * height; i++) {
      host_data[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Reset CUDA context to clear any prior errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Pre-benchmark CUDA error: " << hipGetErrorString(err)
                << std::endl;
    }

    // Test Global Memory
    benchmarkGlobalMemory(host_data.data(), width, height, iterations);

    // Reset CUDA context
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error after global memory benchmark: "
                << hipGetErrorString(err) << std::endl;
    }

    // Test Surface Memory
    benchmarkSurfaceMemory(host_data.data(), width, height, iterations);

    // Reset CUDA context
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error after surface memory benchmark: "
                << hipGetErrorString(err) << std::endl;
    }

    // Test Texture Memory
    benchmarkTextureMemory(host_data.data(), width, height, iterations);

    // Final reset and check
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error after texture memory benchmark: "
                << hipGetErrorString(err) << std::endl;
    }
  }

private:
  void benchmarkGlobalMemory(float *data, int width, int height,
                             int iterations) {
    float *d_input, *d_output;
    hipError_t err;

    // Allocate device memory
    err = hipMalloc(&d_input, width * height * sizeof(float));
    if (err != hipSuccess) {
      std::cerr << "hipMalloc failed for d_input: " << hipGetErrorString(err)
                << std::endl;
      return;
    }
    err = hipMalloc(&d_output, width * height * sizeof(float));
    if (err != hipSuccess) {
      std::cerr << "hipMalloc failed for d_output: " << hipGetErrorString(err)
                << std::endl;
      hipFree(d_input);
      return;
    }

    // Copy data to device
    err = hipMemcpy(d_input, data, width * height * sizeof(float),
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      std::cerr << "hipMemcpy failed for d_input: " << hipGetErrorString(err)
                << std::endl;
      hipFree(d_input);
      hipFree(d_output);
      return;
    }

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run benchmark
    hipEventRecord(start);
    for (int i = 0; i < iterations; i++) {
      gaussianBlurGlobal<<<grid, block>>>(d_input, d_output, width, height);
      std::swap(d_input, d_output);

      err = hipGetLastError();
      if (err != hipSuccess) {
        std::cerr << "Surface memory kernel failed at iteration " << i
                  << hipGetErrorString(err) << std::endl;
        break;
      }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Global memory kernel failed: " << hipGetErrorString(err)
                << std::endl;
    }

    // Measure time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Global Memory: " << milliseconds << " ms" << std::endl;

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void benchmarkSurfaceMemory(float *data, int width, int height,
                              int iterations) {
    SurfaceMemoryManager surface1, surface2;
    hipError_t err;

    // Initialize surfaces
    surface1.initializeSurface2D(width, height);
    surface2.initializeSurface2D(width, height);

    // Copy data to surface
    err = hipMemcpy2DToArray(surface1.getArray(), 0, 0, data,
                              width * sizeof(float), width * sizeof(float),
                              height, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      std::cerr << "hipMemcpy2DToArray failed for surface1: "
                << hipGetErrorString(err) << std::endl;
      return;
    }

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run benchmark
    hipEventRecord(start);
    // gaussianBlurSurface<<<grid, block>>>(surface1.getSurface(),
    //                                      surface2.getSurface(), width,
    //                                      height);
    // if (i % 2 == 0) {
    //   gaussianBlurSurface<<<grid, block>>>(
    //       surface1.getSurface(), surface2.getSurface(), width, height);
    // } else {
    //   gaussianBlurSurface<<<grid, block>>>(
    //       surface2.getSurface(), surface1.getSurface(), width, height);
    // }
    // hipDeviceSynchronize();
    // err = hipGetLastError();
    // if (err != hipSuccess) {
    //   std::cerr << "Surface memory kernel failed at iteration " << ": "
    //             << hipGetErrorString(err) << std::endl;
    //   return;
    // }
    for (int i = 0; i < iterations; i++) {
      gaussianBlurSurface<<<grid, block>>>(
          surface1.getSurface(), surface2.getSurface(), width, height);
      // gaussianBlurSurface<<<grid, block>>>(
      // surface1.getSurface(), surface2.getSurface(), width, height);
      // if (i % 2 == 0) {
      //   gaussianBlurSurface<<<grid, block>>>(
      //       surface1.getSurface(), surface2.getSurface(), width, height);
      // } else {
      //   gaussianBlurSurface<<<grid, block>>>(
      //       surface2.getSurface(), surface1.getSurface(), width, height);
      // }
      err = hipGetLastError();
      if (err != hipSuccess) {
        std::cerr << "Surface memory kernel failed at iteration " << i
                  << hipGetErrorString(err) << std::endl;
        break;
      }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Measure time
    float milliseconds = 0;
    err = hipEventElapsedTime(&milliseconds, start, stop);
    if (err == hipSuccess) {
      std::cout << "Surface Memory: " << milliseconds << " ms" << std::endl;
    } else {
      std::cerr << "hipEventElapsedTime failed for surface memory: "
                << hipGetErrorString(err) << std::endl;
    }

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void benchmarkTextureMemory(float *data, int width, int height,
                              int iterations) {
    hipError_t err;

    // 1. Device memory for output (linear global memory)
    float *d_outputImage;
    err = hipMalloc(&d_outputImage, width * height * sizeof(float));
    if (err != hipSuccess) {
      std::cerr << "hipMalloc failed for d_outputImage: "
                << hipGetErrorString(err) << std::endl;
      return;
    }

    // 2. Create CUDA Array for input
    hipArray *cuArray_input;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    err = hipMallocArray(&cuArray_input, &channelDesc, width, height,
                          hipArrayDefault);
    if (err != hipSuccess) {
      std::cerr << "hipMallocArray failed: " << hipGetErrorString(err)
                << std::endl;
      hipFree(d_outputImage);
      return;
    }

    // 3. Copy host input image to CUDA Array
    err = hipMemcpy2DToArray(cuArray_input, 0, 0, data, width * sizeof(float),
                              width * sizeof(float), height,
                              hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      std::cerr << "hipMemcpy2DToArray failed: " << hipGetErrorString(err)
                << std::endl;
      hipFreeArray(cuArray_input);
      hipFree(d_outputImage);
      return;
    }

    // 4. Create Texture Object
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray_input;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t inputTexObj = 0;
    err = hipCreateTextureObject(&inputTexObj, &resDesc, &texDesc, NULL);
    if (err != hipSuccess) {
      std::cerr << "hipCreateTextureObject failed: " << hipGetErrorString(err)
                << std::endl;
      hipFreeArray(cuArray_input);
      hipFree(d_outputImage);
      return;
    }

    // 5. Set up kernel launch
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 6. Run benchmark
    hipEventRecord(start);
    for (int i = 0; i < iterations; i++) {
      gaussianBlurTexture<<<grid, block>>>(inputTexObj, d_outputImage, width,
                                           height);
      err = hipGetLastError();
      if (err != hipSuccess) {
        std::cerr << "Texture memory kernel failed at iteration " << i << ": "
                  << hipGetErrorString(err) << std::endl;
        break;
      }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 7. Measure time
    float milliseconds = 0;
    err = hipEventElapsedTime(&milliseconds, start, stop);
    if (err == hipSuccess) {
      std::cout << "Texture Memory: " << milliseconds << " ms" << std::endl;
    } else {
      std::cerr << "hipEventElapsedTime failed for texture memory: "
                << hipGetErrorString(err) << std::endl;
    }

    // 8. Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipDestroyTextureObject(inputTexObj);
    hipFreeArray(cuArray_input);
    hipFree(d_outputImage);
  }
};

// ============================================================================
// MAIN DEMONSTRATION
// ============================================================================

int main() {
  // hipArray *testArray;
  // hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  // hipError_t err =
  //     hipMallocArray(&testArray, &channelDesc, 1024, 1024,
  //     hipArrayDefault);
  // if (err != hipSuccess) {
  //   std::cerr << "Test hipMallocArray failed: " << hipGetErrorString(err)
  //             << std::endl;
  // } else {
  //   std::cout << "Test hipMallocArray succeeded!" << std::endl;
  //   hipFreeArray(testArray);
  // }

  std::cout << "CUDA Surface Memory Deep Dive - CUDA 12.0+" << std::endl;
  std::cout << "================================================" << std::endl;

  // Check CUDA capability
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  std::cout << "GPU: " << prop.name << std::endl;
  std::cout << "Compute Capability: " << prop.major << "." << prop.minor
            << std::endl;
  std::cout << "Texture Memory: " << (prop.maxTexture2D[0]) << "x"
            << (prop.maxTexture2D[1]) << std::endl;
  std::cout << std::endl;

  // Run benchmarks
  SurfaceMemoryBenchmark benchmark;
  // benchmark.runBenchmarks(256, 256, 100);
  // benchmark.runBenchmarks(1024, 1024, 100);
  benchmark.runBenchmarks(500, 500, 100);

  std::cout << std::endl;
  std::cout << "=== WHEN TO USE SURFACE MEMORY ===" << std::endl;
  std::cout << "✅ 2D/3D grid computations with neighbor access" << std::endl;
  std::cout << "✅ Image processing and computer vision" << std::endl;
  std::cout << "✅ Scientific simulations (heat, fluid, cellular automata)"
            << std::endl;
  std::cout << "✅ Stencil computations" << std::endl;
  std::cout << "✅ When you need both read and write access to 2D data"
            << std::endl;
  std::cout << std::endl;
  std::cout << "❌ AVOID for linear 1D computations" << std::endl;
  std::cout << "❌ AVOID when atomic operations are critical" << std::endl;
  std::cout << "❌ AVOID for sparse data access patterns" << std::endl;

  return 0;
}
