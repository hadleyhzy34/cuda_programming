#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_surface_types.h>
#include <>
#include <iostream>
#include <vector>

// 5x5 Gaussian kernel
__const__ float kernel[5][5] = {
    {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
    {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
    {7 / 273.0f, 26 / 273.0f, 41 / 273.0f, 26 / 273.0f, 7 / 273.0f},
    {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
    {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}};

// Advanced: Gaussian blur using surface memory
__global__ void gaussianBlurSurface(hipSurfaceObject_t input_surface,
                                    hipSurfaceObject_t output_surface,
                                    int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  // printf("thread execution on %d,%d\n", x, y);

  // // 5x5 Gaussian kernel
  // const float kernel[5][5] = {
  //     {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f},
  //     {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
  //     {7 / 273.0f, 26 / 273.0f, 41 / 273.0f, 26 / 273.0f, 7 / 273.0f},
  //     {4 / 273.0f, 16 / 273.0f, 26 / 273.0f, 16 / 273.0f, 4 / 273.0f},
  //     {1 / 273.0f, 4 / 273.0f, 7 / 273.0f, 4 / 273.0f, 1 / 273.0f}};

  float sum = 0.0f;

  // Convolution with automatic boundary handling
  for (int dy = -2; dy <= 2; dy++) {
    for (int dx = -2; dx <= 2; dx++) {
      if (x + dx >= 0 && x + dx < width && y + dy >= 0 && y + dy < height) {
        float pixel;
        // Surface automatically handles boundary conditions
        surf2Dread(&pixel, input_surface, (x + dx) * sizeof(float), y + dy);
        // sum += pixel * kernel[dy + 2][dx + 2];
        sum += pixel * 0.5;
      }
    }
  }

  surf2Dwrite(sum, output_surface, x * sizeof(float), y);
}

// Simple kernel to scale pixel values using surface memory
__global__ void scaleSurfaceKernel(hipSurfaceObject_t input_surface,
                                   hipSurfaceObject_t output_surface,
                                   int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  printf("handling thread: %d, %d\n", x, y);

  float value;
  surf2Dread(&value, input_surface, x * sizeof(float), y);
  surf2Dwrite(value * 2.0f, output_surface, x * sizeof(float), y);
}

int main() {
  const int width = 64;
  const int height = 64;
  std::cout << "Surface Memory Example: Scaling 64x64 image by 2\n";

  // Initialize host data
  std::vector<float> host_input(width * height, 0.0f);
  std::vector<float> host_output(width * height, 0.0f);
  for (int i = 0; i < width * height; i++) {
    host_input[i] = static_cast<float>(i % 100); // Simple pattern: 0 to 99
  }

  // Allocate CUDA arrays
  hipArray_t input_array, output_array;
  hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float>();
  hipError_t err;

  err = hipMallocArray(&input_array, &channel_desc, width, height,
                        hipArraySurfaceLoadStore);
  if (err != hipSuccess) {
    std::cerr << "hipMallocArray failed for input: " << hipGetErrorString(err)
              << "\n";
    return 1;
  }

  err = hipMallocArray(&output_array, &channel_desc, width, height,
                        hipArraySurfaceLoadStore);
  if (err != hipSuccess) {
    std::cerr << "hipMallocArray failed for output: "
              << hipGetErrorString(err) << "\n";
    hipFreeArray(input_array);
    return 1;
  }

  // Create surface objects
  hipResourceDesc res_desc = {};
  res_desc.resType = hipResourceTypeArray;

  hipSurfaceObject_t input_surface = 0, output_surface = 0;
  res_desc.res.array.array = input_array;
  err = hipCreateSurfaceObject(&input_surface, &res_desc);
  if (err != hipSuccess) {
    std::cerr << "hipCreateSurfaceObject failed for input: "
              << hipGetErrorString(err) << "\n";
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  res_desc.res.array.array = output_array;
  err = hipCreateSurfaceObject(&output_surface, &res_desc);
  if (err != hipSuccess) {
    std::cerr << "hipCreateSurfaceObject failed for output: "
              << hipGetErrorString(err) << "\n";
    hipDestroySurfaceObject(input_surface);
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  // Copy input data to input array
  err = hipMemcpy2DToArray(input_array, 0, 0, host_input.data(),
                            width * sizeof(float), width * sizeof(float),
                            height, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "hipMemcpy2DToArray failed: " << hipGetErrorString(err)
              << "\n";
    hipDestroySurfaceObject(input_surface);
    hipDestroySurfaceObject(output_surface);
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  // Launch kernel
  dim3 block(16, 16);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
  // scaleSurfaceKernel<<<grid, block>>>(input_surface, output_surface, width,
  //                                     height);

  gaussianBlurSurface<<<grid, block>>>(input_surface, output_surface, width,
                                       height);

  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << "\n";
    hipDestroySurfaceObject(input_surface);
    hipDestroySurfaceObject(output_surface);
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << "\n";
    hipDestroySurfaceObject(input_surface);
    hipDestroySurfaceObject(output_surface);
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  // Copy output data back to host
  err = hipMemcpy2DFromArray(host_output.data(), width * sizeof(float),
                              output_array, 0, 0, width * sizeof(float), height,
                              hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr << "hipMemcpy2DFromArray failed: " << hipGetErrorString(err)
              << "\n";
    hipDestroySurfaceObject(input_surface);
    hipDestroySurfaceObject(output_surface);
    hipFreeArray(input_array);
    hipFreeArray(output_array);
    return 1;
  }

  // Verify results
  bool success = true;
  for (int i = 0; i < width * height; i++) {
    if (fabs(host_output[i] - host_input[i] * 2.0f) > 1e-5) {
      std::cerr << "Verification failed at index " << i << ": expected "
                << host_input[i] * 2.0f << ", got " << host_output[i] << "\n";
      success = false;
      break;
    }
  }

  if (success) {
    std::cout << "Surface memory example succeeded! Output verified.\n";
  }

  // Clean up
  hipDestroySurfaceObject(input_surface);
  hipDestroySurfaceObject(output_surface);
  hipFreeArray(input_array);
  hipFreeArray(output_array);

  return success ? 0 : 1;
}
