// host_code_timed.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 1024
#define HEIGHT 1024
#define MASK_WIDTH 5
#define MASK_RADIUS (MASK_WIDTH / 2)

__constant__ float d_mask[MASK_WIDTH * MASK_WIDTH];

// Global memory kernel
__global__ void gaussianBlurGlobal(const float *input, float *output, int width,
                                   int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.0f;

  if (x < width && y < height) {
    for (int j = -MASK_RADIUS; j <= MASK_RADIUS; j++) {
      for (int i = -MASK_RADIUS; i <= MASK_RADIUS; i++) {
        int xi = min(max(x + i, 0), width - 1);
        int yj = min(max(y + j, 0), height - 1);
        sum += input[yj * width + xi] *
               d_mask[(j + MASK_RADIUS) * MASK_WIDTH + (i + MASK_RADIUS)];
      }
    }
    output[y * width + x] = sum;
  }
}

// Texture memory kernel
__global__ void gaussianBlurTexture(hipTextureObject_t texObj, float *output,
                                    int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.0f;

  if (x < width && y < height) {
    for (int j = -MASK_RADIUS; j <= MASK_RADIUS; j++) {
      for (int i = -MASK_RADIUS; i <= MASK_RADIUS; i++) {
        int xi = min(max(x + i, 0), width - 1);
        int yj = min(max(y + j, 0), height - 1);
        sum += tex2D<float>(texObj, xi + 0.5f, yj + 0.5f) *
               d_mask[(j + MASK_RADIUS) * MASK_WIDTH + (i + MASK_RADIUS)];
      }
    }
    output[y * width + x] = sum;
  }
}

// Load Gaussian mask
void loadGaussianMask() {
  float h_mask[MASK_WIDTH * MASK_WIDTH] = {1,  4, 6,  4,  1,  4, 16, 24, 16,
                                           4,  6, 24, 36, 24, 6, 4,  16, 24,
                                           16, 4, 1,  4,  6,  4, 1};
  float sum = 0.0f;
  for (int i = 0; i < MASK_WIDTH * MASK_WIDTH; i++)
    sum += h_mask[i];
  for (int i = 0; i < MASK_WIDTH * MASK_WIDTH; i++)
    h_mask[i] /= sum;
  hipMemcpyToSymbol(HIP_SYMBOL(d_mask), h_mask, sizeof(float) * MASK_WIDTH * MASK_WIDTH);
}

int main() {
  size_t size = WIDTH * HEIGHT * sizeof(float);
  float *h_input = (float *)malloc(size);
  float *h_output = (float *)malloc(size);

  for (int i = 0; i < WIDTH * HEIGHT; i++)
    h_input[i] = rand() % 256;

  float *d_input, *d_output;
  hipMalloc(&d_input, size);
  hipMalloc(&d_output, size);
  hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

  loadGaussianMask();

  dim3 block(2024, 2024);
  dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);

  // =======================
  // Global Memory Timing
  // =======================
  hipEvent_t startGlobal, stopGlobal;
  hipEventCreate(&startGlobal);
  hipEventCreate(&stopGlobal);

  hipEventRecord(startGlobal);
  gaussianBlurGlobal<<<grid, block>>>(d_input, d_output, WIDTH, HEIGHT);
  hipEventRecord(stopGlobal);
  hipEventSynchronize(stopGlobal);

  float timeGlobal;
  hipEventElapsedTime(&timeGlobal, startGlobal, stopGlobal);

  // =======================
  // Texture Memory Setup
  // =======================
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray *cuArray;
  hipMallocArray(&cuArray, &channelDesc, WIDTH, HEIGHT);
  hipMemcpy2DToArray(cuArray, 0, 0, h_input, WIDTH * sizeof(float),
                      WIDTH * sizeof(float), HEIGHT, hipMemcpyHostToDevice);

  struct hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  struct hipTextureDesc texDesc = {};
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  // =======================
  // Texture Memory Timing
  // =======================
  hipEvent_t startTex, stopTex;
  hipEventCreate(&startTex);
  hipEventCreate(&stopTex);

  hipEventRecord(startTex);
  gaussianBlurTexture<<<grid, block>>>(texObj, d_output, WIDTH, HEIGHT);
  hipEventRecord(stopTex);
  hipEventSynchronize(stopTex);

  float timeTex;
  hipEventElapsedTime(&timeTex, startTex, stopTex);

  // =======================
  // Print Results
  // =======================
  printf("Execution Time (Global Memory): %.3f ms\n", timeGlobal);
  printf("Execution Time (Texture Memory): %.3f ms\n", timeTex);

  if (timeTex < timeGlobal) {
    printf("✅ Texture memory is faster by %.3f ms (%.2f%% improvement)\n",
           timeGlobal - timeTex, 100.0f * (timeGlobal - timeTex) / timeGlobal);
  } else {
    printf("❌ Texture memory is slower by %.3f ms (%.2f%% slowdown)\n",
           timeTex - timeGlobal, 100.0f * (timeTex - timeGlobal) / timeGlobal);
  }

  // Cleanup
  hipDestroyTextureObject(texObj);
  hipFreeArray(cuArray);
  hipFree(d_input);
  hipFree(d_output);
  free(h_input);
  free(h_output);

  hipEventDestroy(startGlobal);
  hipEventDestroy(stopGlobal);
  hipEventDestroy(startTex);
  hipEventDestroy(stopTex);

  return 0;
}
