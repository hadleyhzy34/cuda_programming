#include <hip/hip_runtime.h>
#include <iostream>

__global__ void transposeNaive(float *__restrict__ out,
                               const float *__restrict__ in) {
  __shared__ float tile[32][32];
  int x = threadIdx.x;
  int y = threadIdx.y;

  // Load tile from global memory
  tile[y][x] = in[y + x * 32]; // Row-major access (no conflict)

  __syncthreads();

  // Store transposed tile to global memory
  out[y + x * 32] = tile[x][y]; // Column access (causes bank conflict)
}

__global__ void transposePadded(float *__restrict__ out,
                                const float *__restrict__ in) {
  __shared__ float tile[32][33]; // 33 columns (1 extra)
  int x = threadIdx.x;
  int y = threadIdx.y;

  // Load tile from global memory
  tile[y][x] = in[y + x * 32]; // Row-major access (no conflict)

  __syncthreads();

  // Store transposed tile to global memory
  out[y + x * 32] = tile[x][y]; // Column access (no conflict due to padding)
}

void checkCuda(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << msg << " failed: " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main() {
  const int N = 32;
  const int size = N * N * sizeof(float);

  // Host buffers
  float *h_in = new float[N * N];
  float *h_out = new float[N * N];

  // Initialize input matrix
  for (int i = 0; i < N * N; ++i) {
    h_in[i] = i;
  }

  // Device buffers
  float *d_in, *d_out;
  checkCuda(hipMalloc(&d_in, size), "hipMalloc d_in");
  checkCuda(hipMalloc(&d_out, size), "hipMalloc d_out");

  // Copy input to device
  checkCuda(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice),
            "hipMemcpy d_in");

  // Events for timing
  hipEvent_t start, stop;
  checkCuda(hipEventCreate(&start), "hipEventCreate start");
  checkCuda(hipEventCreate(&stop), "hipEventCreate stop");

  // Warm-up
  transposeNaive<<<1, dim3(32, 32)>>>(d_out, d_in);
  hipDeviceSynchronize();

  // Measure naive kernel
  float ms_naive = 0.0f;
  for (int i = 0; i < 100; ++i) {
    hipEventRecord(start);
    transposeNaive<<<1, dim3(32, 32)>>>(d_out, d_in);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms_naive += ms;
  }

  // Measure padded kernel
  float ms_padded = 0.0f;
  for (int i = 0; i < 100; ++i) {
    hipEventRecord(start);
    transposePadded<<<1, dim3(32, 32)>>>(d_out, d_in);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms_padded += ms;
  }

  std::cout << "Naive kernel time: " << ms_naive << " ms" << std::endl;
  std::cout << "Padded kernel time: " << ms_padded << " ms" << std::endl;

  // Cleanup
  delete[] h_in;
  delete[] h_out;
  hipFree(d_in);
  hipFree(d_out);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
