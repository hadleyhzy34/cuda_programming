#include "hip/hip_runtime.h"
#include "sobelFilter.hpp"

// Sobel kernel for edge detection
__global__ void SobelKernel(const float *input, float *output, float *filter,
                            int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col >= width || row >= height)
    return;

  float sum = 0.0f;
  for (int i = -1; i <= 1; i++) {
    for (int j = -1; j <= 1; j++) {
      int imageRow = row + i;
      int imageCol = col + j;
      if (imageRow >= 0 && imageRow < height && imageCol >= 0 &&
          imageCol < width) {
        float pixel = input[imageRow * width + imageCol];
        int filterIdx = (i + 1) * 3 + (j + 1); // 2D to 1D index
        sum += pixel * filter[filterIdx];
      }
    }
  }
  output[row * width + col] = sum;
}

// Template function to launch any CUDA kernel
void SobelFilter::launchKernel(float h_filter[], float *h_input,
                               float *h_output, int width, int height) {

  // Allocate GPU memory
  float *d_input, *d_output;
  hipMalloc(&d_input, sizeof(float) * width * height);
  hipMalloc(&d_output, sizeof(float) * width * height);

  // Copy image data to GPU
  hipMemcpy(d_input, h_input, sizeof(float) * width * height,
             hipMemcpyHostToDevice);

  // allocate sobel filter to global memory
  float *sobelFilter;
  hipMalloc(&sobelFilter, sizeof(float) * 9);

  // copy sobel filter from host code to devicde code
  hipMemcpy(sobelFilter, h_filter, sizeof(float) * 9, hipMemcpyHostToDevice);

  // Launch kernel
  dim3 blockSize(16, 16);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
  SobelKernel<<<gridSize, blockSize>>>(d_input, d_output, sobelFilter, width,
                                       height);

  // Copy result back to host
  hipMemcpy(h_output, d_output, sizeof(float) * width * height,
             hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_input);
  hipFree(d_output);
}
