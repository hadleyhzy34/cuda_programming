#include "hip/hip_runtime.h"
#include "textureSobelFilter.hpp"
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>

// Error checking macro
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at "         \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// Sobel filter in constant memory
__constant__ float sobelFilter[9];

// Sobel kernel for edge detection
__global__ void textureSobelKernel(hipTextureObject_t texObj, float *output,
                                   int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col >= width || row >= height)
    return;

  float sum = 0.0f;
  for (int i = -1; i <= 1; i++) {
    for (int j = -1; j <= 1; j++) {
      float pixel = tex2D<float>(texObj, col + j, row + i);

      sum += pixel * sobelFilter[(i + 1) * 3 + (j + 1)];
    }
  }
  output[row * width + col] = sum;
}

// Template function to launch any CUDA kernel
void Texture::launchKernel(float h_filter[], float *h_input, float *h_output,
                           int width, int height) {
  // Step 1: Allocate CUDA array
  hipArray_t cuArray;
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc<float>(); // Simplified for single float
  CUDA_CHECK(hipMallocArray(&cuArray, &channelDesc, width, height));

  // Step 2: Copy host data to CUDA array
  CUDA_CHECK(hipMemcpy2DToArray(cuArray, 0, 0, h_input, width * sizeof(float),
                                 width * sizeof(float), height,
                                 hipMemcpyHostToDevice));

  // Step 3: Create and configure texture object
  hipResourceDesc resDesc{};
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  hipTextureDesc texDesc{};
  texDesc.addressMode[0] = hipAddressModeClamp; // Clamp for x
  texDesc.addressMode[1] = hipAddressModeClamp; // Clamp for y
  texDesc.filterMode = hipFilterModePoint;      // No interpolation
  texDesc.readMode = hipReadModeElementType;    // Raw float values
  texDesc.normalizedCoords = 0;                  // Use absolute coordinates

  hipTextureObject_t texObj = 0;
  CUDA_CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr));

  // Step 4: Allocate output device memory
  float *d_output;
  CUDA_CHECK(hipMalloc(&d_output, width * height * sizeof(float)));

  // Copy Sobel filter to constant memory
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sobelFilter), h_filter, sizeof(float) * 9));

  // Launch kernel
  dim3 blockSize(16, 16);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);

  textureSobelKernel<<<gridSize, blockSize>>>(texObj, d_output, width, height);

  // Check for any errors during kernel execution
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy result back to host
  CUDA_CHECK(hipMemcpy(h_input, d_output, sizeof(float) * width * height,
                        hipMemcpyDeviceToHost));

  // Cleanup
  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipDestroyTextureObject(texObj));
  CUDA_CHECK(hipFreeArray(cuArray));
}
