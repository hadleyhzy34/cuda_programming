#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//reduction neighbored pairs kernel
__global__ void redunction_neighbored_pairs(int * input, 
	int * temp, int size)
{
	int tid = threadIdx.x;
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

    //local data block pointer
    int * i_data = input + blockDim.x * blockIdx.x;

	if (gid > size)
		return;

	for (int offset = 1; offset <= blockDim.x/2; offset *= 2)
	{
        int index = 2 * offset * tid;
        /*if(index < blockDim.x){
            //input[index] += input[index + offset];
            i_data[index] += i_data[index + offset];
        }*/
        if(gid*offset*2<size){
            //input[gid*offset*2] += input[gid*offset*2 + offset];
            printf("block id is: %d current offset is: %d current thread id is: %d current index is: %d and input value is: %d and %d\n",blockIdx.x, offset, gid, gid*offset*2, input[gid*offset*2], input[gid*offset*2 + offset]);
            input[gid*offset*2] += input[gid*offset*2 + offset];
        }
        //input[gid*offset*2] += input[gid*offset*2+offset]

		/*if (tid % (2 * offset) == 0)
		{
			input[gid] += input[gid + offset];
		}*/
        //synchronize all threads inside one block
		__syncthreads();
	}
    
    //for each block, element that is assigned to the first core/thread of block will be the 
    //sum value of this block
	if (tid == 0)
	{
		temp[blockIdx.x] = input[gid];
        //printf("current block id is: %d, current gid is: %d, temp[%d] = %d\n",blockIdx.x,gid,blockIdx.x,temp[blockIdx.x]);
	}
}

int main(int argc, char ** argv)
{
	printf("Running neighbored pairs reduction kernel \n");
//
	//int size = 1 << 27; //128 Mb of data
	int size = 256;
    int byte_size = size * sizeof(int);
	int block_size = 128;
//
	int * h_input, *h_ref;
	h_input = (int*)malloc(byte_size);
//
    initialize(h_input, size, INIT_RANDOM);
//
//	//get the reduction result from cpu
	int cpu_result = accumulate_cpu(h_input,size);
//
	dim3 block(block_size);
	dim3 grid(size/ block.x);
//
	printf("Kernel launch parameters | grid.x : %d, block.x : %d \n",grid.x, block.x);
//
	int temp_array_byte_size = sizeof(int)* grid.x;
	h_ref = (int*)malloc(temp_array_byte_size);
//
	int * d_input, *d_temp;
//
    hipMalloc((void**)&d_input,byte_size);
    hipMalloc((void**)&d_temp, temp_array_byte_size);
//
    hipMemset(d_temp, 0 , temp_array_byte_size);
    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);
//
    redunction_neighbored_pairs <<<grid, block >>>(d_input,d_temp, size);
//
    hipDeviceSynchronize();
//
    hipMemcpy(h_ref,d_temp, temp_array_byte_size,hipMemcpyDeviceToHost);
//
	int gpu_result = 0;
//
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}
//
//	//validity check
    compare_results(gpu_result, cpu_result);
//
    hipFree(d_temp);
    hipFree(d_input);
//
	free(h_ref);
	free(h_input);
//
    hipDeviceReset();
	return 0;
}
