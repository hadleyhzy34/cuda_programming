#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>


__global__ void sum_array_gpu(int *a,int *b,int *c,int size)
{
    int gid = blockDim.x * blockDim.y * gridDim.x * blockIdx.y + blockDim.x * blockDim.y * blockIdx.x + blockDim.x * threadIdx.y + threadIdx.x;
    if (gid < size)
    {
        c[gid] = a[gid] + b[gid];
    }
    //printf("gid : %d, a[gid] : %d, b[gid] : %d, c[gid] : %d\n", gid, a[gid], b[gid], c[gid]);
}

void sum_array_cpu(int *a, int *b, int *c, int size)
{
    for(int i=0;i<size;i++){
        c[i] = a[i] + b[i];
    }
}

bool checkResult(int *a, int *b, int size)
{
    for(int i=0;i<size;i++){
        if(a[i]!=b[i]){
            printf("the %d th current value of a[i] and b[i] is: %d, %d\n",i,a[i],b[i]);
            return false;
        }
        //printf("the current value of a[i] and b[i] are the same\n");
    }
    return true;
}

int main(int argc, char *argv[])
{	
    int size = 1000;
    
    int dim_x = 2;
    int dim_y = 2;
    int block_x = 16;
    int block_y = 16;
    int byte_size = size * sizeof(int);

    int *a_input,*b_input,*c_output,*gpu_output;
    a_input = (int*)malloc(byte_size);
    b_input = (int*)malloc(byte_size);
    c_output = (int*)malloc(byte_size);
    gpu_output = (int*)malloc(byte_size);

    for(int i=0;i<size;i++)
    {
        a_input[i] = i;
        b_input[i] = i*2;
    }
    
    //cpu matrix sum calculation
    sum_array_cpu(a_input,b_input,c_output,size);


    int * a_gpu_input, * b_gpu_input, *c_gpu_output;
    hipMalloc((void**)&a_gpu_input, byte_size);
    hipMalloc((void**)&b_gpu_input, byte_size);
    hipMalloc((void**)&c_gpu_output, byte_size);

    hipMemcpy(a_gpu_input,a_input,byte_size,hipMemcpyHostToDevice);
    hipMemcpy(b_gpu_input,b_input,byte_size,hipMemcpyHostToDevice);

    dim3 block(block_x,block_y);
    dim3 grid(dim_x,dim_y);

    printf("dimension of each block is: %d, %d\n", block.x, block.y);
    printf("dimension of grid is: %d, %d\n", grid.x, grid.y);
    
    sum_array_gpu<<<grid,block>>>(a_gpu_input,b_gpu_input,c_gpu_output,size);
    hipDeviceSynchronize();
    
    //memory transfer back to host
    hipMemcpy(gpu_output,c_gpu_output,byte_size,hipMemcpyDeviceToHost);

    bool test = checkResult(c_output,gpu_output,size);
    if(test==true){
        printf("the result is true\n");
    }else{
        printf("the result is false\n");
    }

    hipFree(a_gpu_input);
    hipFree(b_gpu_input);
    hipFree(c_gpu_output);
    
    free(a_input);
    free(b_input);
	free(c_output);
    hipDeviceReset();
	
    return 0;
}
