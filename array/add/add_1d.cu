#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>


//handle when there millions of elements for vectors to be processed
__global__ void sum_array_gpu_long(int *a,int *b,int *c,int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //sequentially processing each thread
    while(tid<size){
        c[tid] = a[tid] + b[tid];
        //tid += blockDim.x * gridDim.x;
        if((tid+blockDim.x*gridDim.x)<size){

            printf("max: %d, tid: %d, added_value: %d\n", blockDim.x*gridDim.x, tid, tid+blockDim.x*gridDim.x);
        }
        tid += blockDim.x * gridDim.x;
    }
}


void sum_array_cpu(int *a, int *b, int *c, int size)
{
    for(int i=0;i<size;i++){
        c[i] = a[i] + b[i];
    }
}

bool checkResult(int *a, int *b, int size)
{
    for(int i=0;i<size;i++){
        if(a[i]!=b[i]){
            printf("the %d th current value of a[i] and b[i] is: %d, %d\n",i,a[i],b[i]);
            return false;
        }
        //printf("the current value of a[i] and b[i] are the same\n");
    }
    return true;
}

int main(int argc, char *argv[])
{	
    int size = 100000000;
    printf("size is: %d\n", size);
    int byte_size = size * sizeof(int);

    int *a_input,*b_input,*c_output,*gpu_output;
    a_input = (int*)malloc(byte_size);
    b_input = (int*)malloc(byte_size);
    c_output = (int*)malloc(byte_size);
    gpu_output = (int*)malloc(byte_size);

    for(int i=0;i<size;i++)
    {
        a_input[i] = i;
        b_input[i] = i*2;
    }
    
    //cpu matrix sum calculation
    sum_array_cpu(a_input,b_input,c_output,size);


    int * a_gpu_input, * b_gpu_input, *c_gpu_output;
    hipMalloc((void**)&a_gpu_input, byte_size);
    hipMalloc((void**)&b_gpu_input, byte_size);
    hipMalloc((void**)&c_gpu_output, byte_size);

    hipMemcpy(a_gpu_input,a_input,byte_size,hipMemcpyHostToDevice);
    hipMemcpy(b_gpu_input,b_input,byte_size,hipMemcpyHostToDevice);

    //dim3 block(block_x,block_y);
    //dim3 grid(dim_x,dim_y);
    
    int grid_size = 65535;
    dim3 block(128);
    dim3 grid(grid_size);

    printf("dimension of each block is: %d, %d\n", block.x, block.y);
    printf("dimension of grid is: %d, %d\n", grid.x, grid.y);
    
    sum_array_gpu_long<<<grid,block>>>(a_gpu_input,b_gpu_input,c_gpu_output,size);
    hipDeviceSynchronize();
    
    //memory transfer back to host
    hipMemcpy(gpu_output,c_gpu_output,byte_size,hipMemcpyDeviceToHost);

    bool test = checkResult(c_output,gpu_output,size);
    if(test==true){
        printf("the result is true\n");
    }else{
        printf("the result is false\n");
    }

    hipFree(a_gpu_input);
    hipFree(b_gpu_input);
    hipFree(c_gpu_output);
    
    free(a_input);
    free(b_input);
	free(c_output);
    hipDeviceReset();
	
    return 0;
}
