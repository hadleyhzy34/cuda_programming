#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>


#define gpuErrchk(ans) { gpuAssert((ans),__FILE__,__LINE__);}


//implement one grid with 4 blocks and 256 threads in total, 8x8 threads for each block
__global__ void print_threadIds()
{
	printf("blockIdx,x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, blockDim.z : %d gridDim.x : %d, gridDim.y : %d, gridDim.z : %d \n",blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}


__global__ void unique_idx_calc_threadIdx(int * input)
{
	int tid = threadIdx.x;
    int offset = (blockIdx.x>0)? 4:0;
	printf("blockIdx : %d, threadIdx : %d, value : %d\n", blockIdx.x, tid, input[tid+offset]);
}


__global__ void unique_gid_calculation(int * input){
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int offset = blockIdx.y * gridDim.x * (blockDim.x * blockDim.y) + blockIdx.x * (blockDim.x * blockDim.y);
    //number of threads in one row = gridDim.x * blockDim.x
    //row offset: gridDim.x * blockDim.x * blockIdx.y
    //int offset = blockIdx.x * (blockDim.x * blockDim.y) + blockIdx.y * (blockDim.x * blockDim.y);
    int gid = tid + offset;
    printf("gid: %d, input[gid]: %d \n",gid, input[gid]);
    printf("threadIdx.x : %d, blockIdx.x : %d, blockIdx.y : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d gid : %d value : %d\n", 
           threadIdx.x, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gid, input[gid]);
}


__global__ void mem_trs_test(int * input)
{
    int gid = blockIdx.y * (blockDim.x*blockDim.y)*gridDim.x + blockIdx.x * (blockDim.x*blockDim.y) + threadIdx.x;
    printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
}

__global__ void mem_trs_test1(int * input,int size)
{
    int gid = blockIdx.y * (blockDim.x*blockDim.y)*gridDim.x + blockIdx.x * (blockDim.x*blockDim.y) + threadIdx.x;
    //if(gid<size){
    printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
    //}
}


__global__ void sum_array_gpu(int *a,int *b,int *c,int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size)
    {
        c[gid] = a[gid] + b[gid];
    }
    //printf("gid : %d, a[gid] : %d, b[gid] : %d, c[gid] : %d\n", gid, a[gid], b[gid], c[gid]);
}

void sum_array_cpu(int *a, int *b, int *c, int size)
{
    for(int i=0;i<size;i++){
        c[i] = a[i] + b[i];
    }
}

bool checkResult(int *a, int *b, int size)
{
    for(int i=0;i<size;i++){
        if(a[i]!=b[i]){
            printf("the current value of a[i] and b[i] is: %d, %d",a[i],b[i]);
            return false;
        }
        //printf("the current value of a[i] and b[i] are the same");
    }
    return true;
}

int main()
{	
    int size = 1000;
    //int block_size = 128;
    int byte_size = size * sizeof(int);
    hipError_t error;


    int *a_input,*b_input;
    a_input = (int*)malloc(byte_size);
    b_input = (int*)malloc(byte_size);
    
    int *c_output,*gpu_output;
    c_output = (int*)malloc(byte_size);
    gpu_output = (int*)malloc(byte_size);


    for(int i=0;i<size;i++)
    {
        a_input[i] = i;
        b_input[i] = i*2;
    }
    //cpu matrix sum calculation
    sum_array_cpu(a_input,b_input,c_output,size);

    int * a_gpu_input, * b_gpu_input, *c_gpu_output;
    error = hipMalloc((void**)&a_gpu_input, byte_size);
    if(error != hipSuccess)
    {
        fprintf(stderr,"%s \n", hipGetErrorString(error));
    }

    hipMalloc((void**)&b_gpu_input, byte_size);
    hipMalloc((void**)&c_gpu_output, byte_size);


    hipMemcpy(a_gpu_input,a_input,byte_size,hipMemcpyHostToDevice);
    hipMemcpy(b_gpu_input,b_input,byte_size,hipMemcpyHostToDevice);

    dim3 block(128);
    dim3 grid(8);
    sum_array_gpu<<<grid,block>>>(a_gpu_input,b_gpu_input,c_gpu_output,size);
    hipDeviceSynchronize();


    //memory transfer back to host
    hipMemcpy(gpu_output,c_gpu_output,byte_size,hipMemcpyDeviceToHost);


    //for(int i=0;i<size;i++){
    //    printf("the gpu_output[i] value is: %d",gpu_output[i]);
    //}

    bool test = checkResult(c_output,gpu_output,size);
    if(test==true){
        printf("the result is true");
    }else{
        printf("the result is false");
    }
//    if(checkResult(c_gpu_output,c_output,size)==true){
//        printf("the result is correct");
//    }else{
//        printf("the result is not correct");
//    }

    hipDeviceSynchronize();
    
    hipFree(a_gpu_input);
    hipFree(b_gpu_input);
    hipFree(c_gpu_output);
    free(a_input);
    free(b_input);
	free(c_output);
    hipDeviceReset();
	return 0;
}
