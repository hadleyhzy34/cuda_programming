#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

//implement one grid with 4 blocks and 256 threads in total, 8x8 threads for each block
__global__ void print_threadIds()
{
	printf("blockIdx,x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, blockDim.z : %d gridDim.x : %d, gridDim.y : %d, gridDim.z : %d \n",blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}


int main()
{	
	//define number of threads for each dimension
	int nx,ny,nz;
	nx = 4;
	ny = 4;
	nz = 4;
	
	dim3 block(2,2,2);
	dim3 grid(nx/block.x, ny/block.y, nz/block.z);

	print_threadIds <<< grid, block >>> ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
