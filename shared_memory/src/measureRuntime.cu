#include <hip/hip_runtime.h>

// Function to measure runtime
double measureRuntime(hipFunction_t kernel, void *d_in, void *d_out, int width,
                      int height, int blockSize) {
  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  // Define grid and block dimensions
  dim3 blockDim(blockSize, blockSize);
  dim3 gridDim((width + blockSize - 1) / blockSize,
               (height + blockSize - 1) / blockSize);

  // Launch the kernel
  kernel<<<gridDim, blockDim>>>(d_in, d_out, width, height);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsedTime;
}
