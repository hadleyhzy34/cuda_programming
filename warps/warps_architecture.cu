#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void print_details_of_warps()
{
    int gid = blockIdx.x * gridDim.x * blockDim.x
        + blockIdx.x * blockDim.x + threadIdx.x;

    int warp_id = threadIdx.x / 32;

    int grid_idx = blockIdx.y * gridDim.x + blockIdx.x;

    printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, grid_idx : %d\n",
            threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, grid_idx);
}

int main(int argc, char** argv){
    dim3 block_size(42);
    dim3 grid_size(2,2);

    print_details_of_warps <<< grid_size,block_size>>>();
    hipDeviceSynchronize();

    hipDeviceReset();
    return EXIT_SUCCESS;
}
