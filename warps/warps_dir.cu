#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void print_details_of_warps()
{
    int gid = blockIdx.x * gridDim.x * blockDim.x
        + blockIdx.x * blockDim.x + threadIdx.x;

    int warp_id = threadIdx.x / 32;

    int grid_idx = blockIdx.y * gridDim.x + blockIdx.x;

    printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, grid_idx : %d\n",
            threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, grid_idx);
}


__global__ void code_without_divergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a,b;
    a = b = 0;

    int warp_id = gid / 32;

    if(warp_id % 2 == 0)
    {
        a = 100.0;
        b = 50.0;
    }
    else{
        a = 200;
        b = 75;
    }
}

__global__ void divergence_code()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float a,b;
    a = b = 0;

    if(gid%2 == 0){
        a = 100.0;
        b = 50.0;
    }else{
        a = 200;
        b = 75;
    }
}

int main(int argc, char** argv){
    int size = 1 << 22;

    dim3 block_size(128);
    dim3 grid_size((size+block_size.x-1)/block_size.x);

    code_without_divergence <<< grid_size, block_size>>>();
    hipDeviceSynchronize();

    divergence_code <<< grid_size, block_size>>>();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}
